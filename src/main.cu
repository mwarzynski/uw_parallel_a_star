#include "hip/hip_runtime.h"
#include "a_star.cuh"

__device__ void memory_init(Memory* memory, size_c mem_size) {
    memory->data = (void*)((size_c)memory + sizeof(Memory));
    memory->size = mem_size - sizeof(Memory);
    memory->allocated = 0;
}

__device__ void* memory_allocate(size_t size) {
    size_c allocated = atomicAdd(&(p.memory->allocated), size);
    assert(allocated + size < p.memory->size);
    return (void*)((size_c)p.memory->data + allocated);
}

__device__ int node_id_puzzle(NodePuzzle* node) {
    int count = p.puzzle.n * p.puzzle.n;
    int id = 0;
    for (int i = 0; i < count; i++) {
        id += i*node->numbers[i];
    }
    return id;
}

__device__ int node_id_pathfinding(NodePathfinding* node) {
    return node->x*p.pathfinding.dim_y + node->y;
}

__device__ int node_id(Node* node) {
    void *nd = node_data(node);
    switch (p.type) {
        case PROBLEM_TYPE_PUZZLE:
            return node_id_puzzle((NodePuzzle*)nd);
        case PROBLEM_TYPE_PATHFINDING:
            return node_id_pathfinding((NodePathfinding*)nd);
        default:
            assert(false);
            return -1;
    }
}

__device__ void* node_data(Node* node) {
    return (void*)(node + sizeof(Node));
}

__device__ size_t node_size() {
    switch (p.type) {
        case PROBLEM_TYPE_PUZZLE:
            return sizeof(Node) + sizeof(int)*p.puzzle.n*p.puzzle.n;
        case PROBLEM_TYPE_PATHFINDING:
            return sizeof(Node) + sizeof(NodePathfinding);
        default:
            assert(false);
            return 0;
    }
}

__device__ void queues_init(Queue *queues, size_c all_memory, int k) {
    size_c queues_memory = sizeof(Queue)*k;
    size_c items_memory = (all_memory - queues_memory) / k;
    for (int i = 0; i < k; i++) {
        queues[i].count = 0;
        queues[i].capacity = items_memory / sizeof(Node*);
        queues[i].items = (Node*)((size_c)queues + queues_memory +
                (i*items_memory));
    }
}

__device__ void queue_push(Queue *queue, Node* node) {
    assert(queue->count < queue->capacity);
    queue->count++;
    int i = queue->count-1;
    int p;
    while (i > 0) {
        p = (i-1)/2;
        if (queue->items[p].f < node->f) {
            break;
        }
        queue->items[i] = queue->items[p];
        i = p;
    }
    queue->items[i] = *node;
}

__device__ void queue_downify(Queue *queue, int i) {
    int l = 2*i+1;
    int r = l+1;
    // Determine if we need to push value down.
    int min = i;
    if (l < queue->count && queue->items[l].f < queue->items[min].f) {
        min = l;
    }
    if (r < queue->count && queue->items[r].f < queue->items[min].f) {
        min = r;
    }
    // If one of our children has a better value, bring it up.
    // We also need to make sure our subtree will have correct values.
    if (min != i) {
        Node t = queue->items[i];
        queue->items[i] = queue->items[min];
        queue->items[min] = t;
        queue_downify(queue, min);
    }
}

__device__ void queue_pop(Queue *queue, Node *result) {
    assert(queue->count > 0);
    *result = queue->items[0];
    queue->items[0] = queue->items[--queue->count];
    queue_downify(queue, 0);
}

__device__ int map_hash(Map* map, int j, Node *node) {
    return 0;
}

__device__ void map_init(Map *map, size_c map_size) {
    map->hs = MAP_HASHING_FUNCTIONS;
    map->nodes = (Node**)((size_c)map + sizeof(Map));
    map->nodes_count = (map_size - sizeof(Map)) / sizeof(Node**);
}

__device__ void map_deduplicate(Node* nodes, Node* nodes_dest, int n) {

}

__global__ void gpu_astar_init(Problem hp) {
    p = hp;
    memory_init(p.memory, p.mem_size);
    queues_init(p.queues, p.queues_size, p.k);
    map_init(p.map, p.map_size);
}

__global__ void gpu_astar(int k) {

}

int main() {
    Problem p;
    p.mem_size = 1024*1024 * 9L;
    p.queues_size = 1024 * 512L;
    p.map_size = 1024 * 512L;
    p.k = 2;

    p.puzzle.n = 3;
    p.type = PROBLEM_TYPE_PUZZLE;

    // Initialize memory.
    handleError(hipMalloc((void**)&p.memory, p.mem_size));
    handleError(hipMalloc((void**)&p.queues, p.queues_size));
    handleError(hipMalloc((void**)&p.map, p.map_size));
    gpu_astar_init<<<1, 1>>>(p);
    hipDeviceSynchronize();

    // Run algorithm.
    gpu_astar<<<1, 1>>>(2);
    hipDeviceSynchronize();

    // TODO: Fetch results from GPU.

    // Free memory.
    handleError(hipFree(p.memory));
    handleError(hipFree(p.queues));
    handleError(hipFree(p.map));
    hipDeviceSynchronize();

    return 0;
}

